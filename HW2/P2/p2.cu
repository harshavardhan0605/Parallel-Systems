/*Single Author info:
mreddy2 Muppidi Harshavardhan Reddy */

/* Program to compute Pi using Monte Carlo methods */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#define SEED 35791246
#define THREADS 512

__global__ void integrate(double *x_d,double *y_d, int nitter, hiprandState *state,double *gsum)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int i;
    hiprand_init(SEED, idx, 0, &state[idx]);        //Initializing for Currand Function
    __shared__ double sum[THREADS];
    if(idx<nitter)
    {
        hiprandState localState = state[idx];
        x_d[idx] = hiprand_uniform(&localState);
        y_d[idx] = hiprand_uniform(&localState);
        if((x_d[idx]*x_d[idx] + y_d[idx]*y_d[idx])<=1){
        sum[threadIdx.x] = 1;}
        else 
        {
            sum[threadIdx.x] =0;}
    }
    // block reduction
  __syncthreads();
  for (i = blockDim.x / 2; i > 0; i >>= 1) { /* per block */
    if (threadIdx.x < i)
      sum[threadIdx.x] += sum[threadIdx.x + i];
    __syncthreads();
  }
  if (threadIdx.x == 0){
    gsum[blockIdx.x] = sum[threadIdx.x];     // Getting Each Block Total Points 
}
}

int main(int argc, char** argv)
{
   int niter=0;                     // Total Number of Points 
   double *x_d,*y_d, *z,*result_d;  // Device Copy
   int *blocks_d; 
   double count, pi;                // Host Copies
   int i,blocks;  
   hiprandState *states_d;           // For Currand State Device Copy
    
   niter = atoi(argv[1]);
   blocks = (niter/THREADS) + 1;    // Caluclating Number of Blocks Based on total Points
   
   z = (double *)malloc(niter * sizeof(double));    // Allocating Memory in CPU to use for copying back from GPU

    // GPU Memory Allocation for device copies
   hipMalloc( (void **) &blocks_d, sizeof(int) * 1 );
   hipMalloc((void **)&states_d, niter * sizeof(hiprandState));
   hipMalloc((void **)&x_d, niter * sizeof(double));
	hipMalloc((void **)&y_d, niter * sizeof(double));
    hipMalloc( (void **) &result_d, sizeof(double) * THREADS * blocks);

    integrate<<<blocks,THREADS>>>(x_d, y_d, niter,states_d,result_d);
   
    // copy back from GPU to CPU
    hipMemcpy( z, result_d, blocks*sizeof(double), hipMemcpyDeviceToHost);

    for(i=0;i<blocks;i++)       // Summing up total points at all Blocks
    {
        count+= z[i];
    }

    hipFree(x_d);              // Deallocation of the Memory
    hipFree(blocks_d);
    hipFree(y_d);
    hipFree(result_d);
    hipFree(states_d);

    pi=(double)count/niter*4;
    printf(" # of trials= %d , estimate of pi is %.16f \n",niter,pi);
}
